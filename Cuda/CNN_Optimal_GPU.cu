#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <filesystem>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

#define FILTER_SIZE    3
#define BLOCK_SIZE     16
#define POOL_SIZE      3
#define IMG_PATH           "data/training_set/cats/"
#define IMG_PATH_FINAL     "data/gpu_optimal_output/cats/"

// Filter stored in constant memory (flattened)
__constant__ int d_filter[FILTER_SIZE * FILTER_SIZE];

// Fused convolution + max-pooling kernel
extern "C" __global__ void convPoolKernel(
    const unsigned char* __restrict__ input,
    unsigned char* __restrict__ output,
    int width, int height,
    int pooledW, int pooledH)
{
    const int pad = FILTER_SIZE / 2;
    const int tileW = BLOCK_SIZE * POOL_SIZE + FILTER_SIZE - 1;
    extern __shared__ unsigned char tile[];

    int outX = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int outY = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int baseX = blockIdx.x * BLOCK_SIZE * POOL_SIZE;
    int baseY = blockIdx.y * BLOCK_SIZE * POOL_SIZE;

    for (int ty = threadIdx.y; ty < tileW; ty += BLOCK_SIZE) {
        for (int tx = threadIdx.x; tx < tileW; tx += BLOCK_SIZE) {
            int gx = baseX + tx - pad;
            int gy = baseY + ty - pad;
            unsigned char val = 0;
            if (gx >= 0 && gx < width && gy >= 0 && gy < height) {
                val = input[gy * width + gx];
            }
            tile[ty * tileW + tx] = val;
        }
    }
    __syncthreads();

    if (outX < pooledW && outY < pooledH) {
        int maxVal = 0;
        for (int py = 0; py < POOL_SIZE; ++py) {
            for (int px = 0; px < POOL_SIZE; ++px) {
                int sum = 0;
                int startY = threadIdx.y * POOL_SIZE + py;
                int startX = threadIdx.x * POOL_SIZE + px;
                for (int fy = 0; fy < FILTER_SIZE; ++fy) {
                    for (int fx = 0; fx < FILTER_SIZE; ++fx) {
                        unsigned char pix = tile[(startY + fy) * tileW + (startX + fx)];
                        sum += pix * d_filter[fy * FILTER_SIZE + fx];
                    }
                }
                maxVal = max(maxVal, sum);
            }
        }
        output[outY * pooledW + outX] = static_cast<unsigned char>(maxVal);
    }
}

int main(int argc, char** argv) {
    // ===== CUDA Device Information =====
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA Device Information:" << std::endl;
    std::cout << "-----------------------" << std::endl;
    std::cout << "Number of CUDA devices: " << deviceCount << std::endl << std::endl;
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev);
        std::cout << "Device " << dev << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total global memory: " << (prop.totalGlobalMem / (1024 * 1024)) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max threads dimensions: ("
                  << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", "
                  << prop.maxThreadsDim[2] << ")" << std::endl;
    }
    std::cout << "-----------------------" << std::endl << std::endl;

    // Ensure output directory exists
    std::filesystem::create_directories(IMG_PATH_FINAL);

    // Read filter on host
    int h_filter[FILTER_SIZE * FILTER_SIZE] = { /* your 3x3 values */ };
    hipMemcpyToSymbol(HIP_SYMBOL(d_filter), h_filter, sizeof(h_filter));

    // Prepare pinned host buffers
    unsigned char *h_input, *h_output;
    hipHostAlloc(&h_input,  1920 * 1080 * sizeof(unsigned char), hipHostMallocDefault);
    hipHostAlloc(&h_output, 1920 * 1080 * sizeof(unsigned char), hipHostMallocDefault);

    // Create CUDA streams
    const int STREAM_COUNT = 4;
    std::vector<hipStream_t> streams(STREAM_COUNT);
    for (int i = 0; i < STREAM_COUNT; ++i) hipStreamCreate(&streams[i]);

    // Device buffers
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input,  1920 * 1080 * sizeof(unsigned char));
    hipMalloc(&d_output, 1920 * 1080 / (POOL_SIZE * POOL_SIZE) * sizeof(unsigned char));

    // List images
    std::vector<std::filesystem::path> files;
    for (auto& p : std::filesystem::directory_iterator(IMG_PATH))
        files.push_back(p.path());

    // Timing
    hipEvent_t startE, endE;
    hipEventCreate(&startE);
    hipEventCreate(&endE);
    hipEventRecord(startE);

    int idx = 0;
    for (auto& file : files) {
        hipStream_t s = streams[idx % STREAM_COUNT];
        cv::Mat img = cv::imread(file.string(), cv::IMREAD_GRAYSCALE);
        int W = img.cols, H = img.rows;
        int pooledW = (W + POOL_SIZE - 1) / POOL_SIZE;
        int pooledH = (H + POOL_SIZE - 1) / POOL_SIZE;

        memcpy(h_input, img.data, W * H * sizeof(unsigned char));
        hipMemcpyAsync(d_input, h_input, W * H * sizeof(unsigned char), hipMemcpyHostToDevice, s);

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((pooledW + BLOCK_SIZE - 1)/BLOCK_SIZE, (pooledH + BLOCK_SIZE - 1)/BLOCK_SIZE);
        size_t sharedBytes = (BLOCK_SIZE * POOL_SIZE + FILTER_SIZE - 1) *
                             (BLOCK_SIZE * POOL_SIZE + FILTER_SIZE - 1) * sizeof(unsigned char);
        convPoolKernel<<<grid, block, sharedBytes, s>>>(d_input, d_output, W, H, pooledW, pooledH);

        hipMemcpyAsync(h_output, d_output, pooledW * pooledH * sizeof(unsigned char), hipMemcpyDeviceToHost, s);
        hipStreamSynchronize(s);

        cv::Mat outImg(pooledH, pooledW, CV_8UC1, h_output);
        std::string output_filename = "final_" + file.filename().string();
        cv::imwrite(std::string(IMG_PATH_FINAL) + output_filename, outImg);
        idx++;
    }

    hipEventRecord(endE);
    hipEventSynchronize(endE);
    float ms;
    hipEventElapsedTime(&ms, startE, endE);

    // ===== Processing Summary =====
    std::cout << "Processing complete!" << std::endl;
    std::cout << "=================================" << std::endl;
    std::cout << "Total images processed: " << files.size() << std::endl;
    std::cout << "Failed to process: 0" << std::endl;
    std::cout << "Total processing time: " << static_cast<int>(ms) << " ms" << std::endl;
    std::cout << "=================================" << std::endl;

    // Cleanup
    for (auto& s : streams) hipStreamDestroy(s);
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
